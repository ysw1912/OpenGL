#include "hip/hip_runtime.h"
#include "convexhull4.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>

struct ComparePointByX
{
	__host__ __device__ bool operator()(const Point lhs, const Point rhs) const
	{
		return lhs.x < rhs.x;
	}
};

// �߶�d_Points[i]d_Points[j]��ĳһ��side
__global__ void UpdateState(Point* d_Points, uint32_t* d_State,
	uint32_t i, uint32_t j)
{
	uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	int16_t v = Orientation(d_Points[i], d_Points[j], d_Points[idx]);
	d_State[idx] = 0 * (v == -1) + 1 * (v == 1);
}

void ConvexHull4(vector<Point>& points, vector<Point>& hull)
{
	size_t size = points.size();

	Point* d_Points;
	checkCudaError(hipMalloc((void**)&d_Points, size * sizeof(Point)));
	checkCudaError(hipMemcpy(d_Points, points.data(), size * sizeof(Point), hipMemcpyHostToDevice));

	// Step 1: ȡxֵ������С���λ��
	thrust::device_vector<Point> t_Points(d_Points, d_Points + size);
	thrust::device_vector<Point>::iterator it;
	it = thrust::max_element(t_Points.begin(), t_Points.end(), ComparePointByX());
	size_t maxPos = it - t_Points.begin();
	it = thrust::min_element(t_Points.begin(), t_Points.end(), ComparePointByX());
	size_t minPos = it - t_Points.begin();

	hull.push_back(points[maxPos]);
	hull.push_back(points[minPos]);

	// ������ʼ�Ķ���d_SegHead����, ����1��Ԫ��Ϊ1, ����ȫ0
	// ������ʼ��״̬d_State����
	uint32_t *h_SegHead = new uint32_t[size], *h_State = new uint32_t[size];
	memset(h_SegHead, 0, size * sizeof(uint32_t));
	memset(h_State, 0, size * sizeof(uint32_t));
	h_SegHead[0] = 1;

	uint32_t *d_SegHead, *d_State;
	checkCudaError(hipMalloc((void**)&d_SegHead, size * sizeof(uint32_t)));
	checkCudaError(hipMalloc((void**)&d_State, size * sizeof(uint32_t)));
	checkCudaError(hipMemcpy(d_SegHead, h_SegHead, size * sizeof(uint32_t), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(d_State, h_State, size * sizeof(uint32_t), hipMemcpyHostToDevice));

	UpdateState<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_Points, d_State, minPos, maxPos);


	// �ͷ��ڴ�
	delete[] h_SegHead;
	delete[] h_State;

	checkCudaError(hipFree(d_SegHead));
	checkCudaError(hipFree(d_State));
	checkCudaError(hipFree(d_Points));
}